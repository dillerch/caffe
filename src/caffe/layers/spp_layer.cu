#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#ifdef __CDT_PARSER__
#define __global__
#define CUDA_KERNEL_LOOP(i, n) for (int i = 0; i < (n); ++i)
#endif

namespace caffe {

template<typename Dtype>
__global__ void SPPForward(const int num_threads,
    const Dtype* const bottom_data, Dtype* const top_data, int* const mask,
    const int bottom_width, const int bottom_height,
    const int num_bins_w, const int num_bins_h, const int total_num_bins,
    const Dtype bin_size_w, const Dtype bin_size_h,
    const int channels, const int previous_bins) {
  //Run a CUDA kernel loop; grid stride looping
  CUDA_KERNEL_LOOP(index, num_threads) {
    //Get current position
    const int n = index / num_bins_w / num_bins_h / channels;
    const int c = (index / num_bins_w / num_bins_h) % channels;
    const int nbh = (index / num_bins_w) % num_bins_h;
    const int nbw = index % num_bins_w;

    //Calculate bin start and end coordinates
    const int wstart = max(static_cast<int>(floor(static_cast<Dtype>(nbw) * bin_size_w)), 0);
    const int hstart = max(static_cast<int>(floor(static_cast<Dtype>(nbh) * bin_size_h)), 0);
    const int wend = min(static_cast<int>(ceil(static_cast<Dtype>(nbw + 1) * bin_size_w)), bottom_width);
    const int hend = min(static_cast<int>(ceil(static_cast<Dtype>(nbh + 1) * bin_size_h)), bottom_height);

    //Max val and idx in registers
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;

    //Calculate current pointer position
    const Dtype* const bottom_slice = bottom_data + (n * channels + c) * bottom_height * bottom_width;

    //Max pooling
    for(int h = hstart; h < hend; ++h) {
      for(int w = wstart; w < wend; ++w) {
        const int pos = h * bottom_width + w;
        if(bottom_slice[pos] > maxval) {
          maxval = bottom_slice[pos];
          maxidx = pos;
        }
      }
    }
    //Write results to global memory
    const int bin_index = (n * channels + c) * total_num_bins + previous_bins + nbh * num_bins_w + nbw;
    top_data[bin_index] = maxval;
    mask[bin_index] = maxidx;
  }
}

template<typename Dtype>
void SPPLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top) {
    //Get top, bottom and mask data
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int* mask = max_idx_.mutable_gpu_data();

    //Store previous pyramid bins
    int previous_bins = 0;

    //Loop over pyramid layers
    for(int p_layer = 0; p_layer < pyramid_height_; ++p_layer) {
      //Calculate bin width and height
      Dtype bin_size_w = static_cast<Dtype>(bottom_w_) / static_cast<Dtype>(num_bins_w_[p_layer]);
      Dtype bin_size_h = static_cast<Dtype>(bottom_h_) / static_cast<Dtype>(num_bins_h_[p_layer]);

      //The number of workers we will spawn on the GPU instead of looping over num, channels and bins
      const int count = num_ * channels_ * num_bins_w_[p_layer] * num_bins_h_[p_layer];

      //Launch CUDA kernel
      SPPForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count,
        bottom_data, top_data, mask,
        bottom_w_, bottom_h_,
        num_bins_w_[p_layer], num_bins_h_[p_layer], total_num_bins_,
        bin_size_w, bin_size_h,
        channels_, previous_bins);

      //Update previous bins
      previous_bins += num_bins_h_[p_layer] * num_bins_w_[p_layer];
    }

    CUDA_POST_KERNEL_CHECK;
}

template<typename Dtype>
__global__ void SPPBackward(const int num_threads,
    Dtype* const bottom_diff, const Dtype* const top_diff, const int* const mask,
    const int bottom_width, const int bottom_height,
    const int num_bins_w, const int num_bins_h, const int total_num_bins,
    const Dtype bin_size_w, const Dtype bin_size_h,
    const int channels, const int previous_bins) {
  //Run a CUDA kernel loop; grid stride looping
  CUDA_KERNEL_LOOP(index, num_threads) {
    //Get current position
    const int n = index / bottom_width / bottom_height / channels;
    const int c = (index / bottom_width / bottom_height) % channels;
    const int h = (index / bottom_width) % bottom_height;
    const int w = index % bottom_width;

    //Calculate bin start and end coordinates
    const int wstart = max(static_cast<int>(floor(static_cast<Dtype>(w) / bin_size_w)), 0);
    const int hstart = max(static_cast<int>(floor(static_cast<Dtype>(h) / bin_size_h)), 0);
    const int wend = min(static_cast<int>(ceil(static_cast<Dtype>(w) / bin_size_w + 1)), num_bins_w);
    const int hend = min(static_cast<int>(ceil(static_cast<Dtype>(h) / bin_size_h + 1)), num_bins_h);

    //Calculate current pointer position
    const int offset = (n * channels + c) * total_num_bins;
    const Dtype* const top_diff_slice = top_diff + offset;
    const int* const mask_slice = mask + offset;

    //Gradient in register
    Dtype gradient = 0;

    //Accumulate gradient
    for (int ph = hstart; ph < hend; ++ph)
      for (int pw = wstart; pw < wend; ++pw)
        if (mask_slice[previous_bins + ph * num_bins_w + pw] == h * bottom_width + w)
          gradient += top_diff_slice[previous_bins + ph * num_bins_w + pw];

    //Write gradient to global memory
    bottom_diff[index] = gradient;
  }
}
	
template<typename Dtype>
void SPPLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if(!propagate_down[0]) return;

  //Initialize bottom diff
  caffe_gpu_set(bottom[0]->count(), Dtype(0.), bottom[0]->mutable_gpu_diff());

  //Get top diff, bottom diff and mask
  const Dtype* const top_diff = top[0]->gpu_diff();
  Dtype* const bottom_diff = bottom[0]->mutable_gpu_diff();
  const int* const mask = max_idx_.gpu_data();

  //Store previous pyramid bins
  int previous_bins = 0;

  for(int p_layer = 0; p_layer < pyramid_height_; ++p_layer) {
    //The number of workers we will spawn on the GPU: One per bottom data point
    const int count = bottom[0]->count();

    //Calculate bin width and height
    Dtype bin_size_w = static_cast<Dtype>(bottom_w_) / static_cast<Dtype>(num_bins_w_[p_layer]);
    Dtype bin_size_h = static_cast<Dtype>(bottom_h_) / static_cast<Dtype>(num_bins_h_[p_layer]);

    //Launch CUDA kernel
    SPPBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count,
        bottom_diff, top_diff, mask,
        bottom_w_, bottom_h_,
        num_bins_w_[p_layer], num_bins_h_[p_layer], total_num_bins_,
        bin_size_w, bin_size_h,
        channels_, previous_bins);

    //Update previous bins
    previous_bins += num_bins_h_[p_layer] * num_bins_w_[p_layer];
  }
  CUDA_POST_KERNEL_CHECK;
}
	
	INSTANTIATE_LAYER_GPU_FUNCS(SPPLayer);
}

