
#include <hip/hip_runtime.h>
/*#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#ifdef __CDT_PARSER__
#define __global__
#define CUDA_KERNEL_LOOP(a, b)
#endif

namespace caffe {

template<typename Dtype>
__global__
void SPPForward(const int num_threads,
  const Dtype* const bottom_data, Dtype* const top_data, int* mask,
  const int bottom_width, const int bottom_height,
  const int num_bins_w, const int num_bins_h,
  const Dtype bin_size_w, const Dtype bin_size_h,
  const int channels, const int previous_bins) {
    //Run a CUDA kernel loop
    CUDA_KERNEL_LOOP(index, num_threads) {
      //Get current position
      const int n = index / num_bins_w / num_bins_h / channels;
      const int c = (index / num_bins_w / num_bins_h) % channels;
      const int nbh = (index / num_bins_w) % num_bins_h;
      const int nbw = index % num_bins_w;

      //Calculate start and end
      int hstart = max(static_cast<int>(floor(static_cast<Dtype>(nbh) * bin_size_h)), 0);
      int wstart = max(static_cast<int>(floor(static_cast<Dtype>(nbw) * bin_size_w)), 0);
      int hend = min(static_cast<int>(ceil(static_cast<Dtype>(nbh + 1) * bin_size_h)), bottom_height);
      int wend = min(static_cast<int>(ceil(static_cast<Dtype>(nbw + 1) * bin_size_w)), bottom_width);

      Dtype maxval = -FLT_MAX;
      int maxidx = -1;
      const Dtype* const bottom_slice = bottom_data + (n * channels + c) * bottom_width * bottom_height;
      for(int h = hstart; h < hend; ++h) {
        for(int w = wstart; w < wend; ++w) {
          if(bottom_slice[h * bottom_width + w] > maxval) {
            maxidx = h * bottom_width + w;
            maxval = bottom_slice[maxidx];
          }
        }
      }
      top_data[previous_bins + index] = maxval;
      mask[previous_bins + index] = maxidx;
  }
}

template <typename Dtype>
void SPPLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top) {
    //Get top, bottom and mask data
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int* mask = max_idx_.mutable_gpu_data();

    int previous_bins = 0;
    //Loop over pyramid layers
    for(int i = 0; i < pyramid_height_; ++i) {
      const int count = num_bins_w_[i] * num_bins_h_[i] * num_ * channels_;
      Dtype bin_size_w = static_cast<Dtype>(bottom_w_) / static_cast<Dtype>(num_bins_w_[p_layer]);
      Dtype bin_size_h = static_cast<Dtype>(bottom_h_) / static_cast<Dtype>(num_bins_h_[p_layer]);

      SPPForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count,
        bottom_data, top_data, mask,
        bottom_w_, bottom_h_,
        num_bins_w_[i], num_bins_h_[i],
        bin_size_w, bin_size_h,
        channels_, previous_bins);

      previous_bins += num_bins_h_[p_layer] * num_bins_w_[p_layer];
    }
    CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__
void SPPBackward(const int num_threads,
    const Dtype* const bottom_diff, Dtype* const top_diff, int* mask,
    const int bottom_width, const int bottom_height,
    const int num_bins_w, const int num_bins_h,
    const Dtype bin_size_w, const Dtype bin_size_h,
    const int channels, const int previous_bins) {
  //Run a CUDA kernel loop
  CUDA_KERNEL_LOOP(index, num_threads) {
    //Get current position
    const int n = index / bottom_width / bottom_height / channels;
    const int c = (index / bottom_width / bottom_height) % channels;
    const int h = (index / bottom_width) % bottom_height;
    const int w = index % bottom_width;

    const int phstart = (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    const int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    const int pwstart = (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    const int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    const int offset = (n * channels + c) * bin_size_h * bin_size_w;
    const Dtype* const top_diff_slice = top_diff + offset;
    const int* const mask_slice = mask + offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask_slice[ph * pooled_width + pw] == h * width + w) {
          gradient += top_diff_slice[ph * pooled_width + pw];
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}
	
template <typename Dtype>
void SPPLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) return;
  //Get top and bottom diff
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  const int* mask = max_idx_.gpu_data();

  int previous_bins = 0;
  for(int i = 0; i < pyramid_height_; ++i) {
    SPPBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count,
        bottom_diff, top_diff, mask,
        bottom_w_, bottom_h_,
        num_bins_w_[i], num_bins_h_[i],
        bin_size_w, bin_size_h,
        channels_, previous_bins);
    previous_bins += num_bins_h_[p_layer] * num_bins_w_[p_layer];
  }
  CUDA_POST_KERNEL_CHECK;
}
	
	INSTANTIATE_LAYER_GPU_FUNCS(SPPLayer);
}*/
